#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <fstream>
#include <sstream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include <stdbool.h>
#include <math.h>
using namespace cv;
using namespace std;

#define BLOCKSIZE 32 //The number of threads per block should be not greater than 1024

#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


typedef struct
{
    float x,y,w,h,s;

}box;

__device__
float IOUcalc(box b1, box b2)
{
    float ai = (float)(b1.w+1)*(b1.h+1); //why plus 1??
    float aj = (float)(b2.w+1)*(b2.h+1);
    float x_inter, x2_inter, y_inter, y2_inter;

    x_inter = max(b1.x,b1.x);
    y_inter = max(b1.y,b2.y);

    x2_inter = min((b1.x + b1.w),(b2.x + b2.w));
    y2_inter = min((b1.y + b1.h),(b2.y + b2.h));

    float w = (float)max((float)0, x2_inter - x_inter + 1);  
    float h = (float)max((float)0, y2_inter - y_inter + 1);  

    float inter = ((w*h)/(ai + aj - w*h));
    return inter;
}



__global__
void NMS_GPU(box *d_b, bool *d_res)
{
    int abs_y = (blockIdx.y * blockDim.y) + threadIdx.y;
    int abs_x = (blockIdx.x * blockDim.x) +threadIdx.x;


    float theta = 0.5;


    if(d_b[abs_x].s < d_b[abs_y].s)
    {
        if(IOUcalc(d_b[abs_y],d_b[abs_x])>theta)
        {
            d_res[abs_x] = false; 
        }
    }
  
}





int main()
{
    int count = 75;

    bool *h_res =(bool *)malloc(sizeof(bool)*count);

    for(int i=0; i<count; i++)
    {
        h_res[i] = true;
    }


    box b[count];
    
    std::ifstream in;
    std::string line;
    
    in.open("input_box_dssa56_75.txt"); //x1, y1, w, h
    if (in.is_open()) 
    {
        int i = 0;
        while(getline(in, line))
        {
            istringstream iss(line);
            iss >> b[i].x;
            iss >> b[i].y;
            iss >> b[i].w;
            iss >> b[i].h;
            i+=1;
            if(i==count) break;
        }
    }
    in.close();
    
    in.open("sorted_indices_75.txt");
    if (in.is_open()) 
    {
        int i = 0;
        int cur = -1;
        while(in >> cur)
        {
            b[cur].s = 1 - 0.01*i;
            i+=1;
            if(i==count) break;
        }
    }
    in.close();
    
    box *d_b;
    bool *d_res;

    gpuErrchk(hipMalloc((void**)&d_res, count*sizeof(bool)));
    gpuErrchk(hipMemcpy(d_res, h_res,sizeof(bool)*count, hipMemcpyHostToDevice));

    gpuErrchk(hipMalloc((void**)&d_b,sizeof(box)*count));
    gpuErrchk(hipMemcpy(d_b, b,sizeof(box)*count, hipMemcpyHostToDevice));
    
    //Setting 1: can only work when count <= 1024
    //NMS_GPU<<<dim3(1,count,1),count>>>(d_b,d_res);
    
    //Setting 2: work when count > 1024
    //NMS_GPU<<<dim3(count,count,1), 1>>>(d_b,d_res);
    
    //Setting 3: work when count > 1024, faster than Setting 2
    dim3 gridSize(int(ceil(float(count)/BLOCKSIZE)), int(ceil(float(count)/BLOCKSIZE)),1);
    dim3 blockSize(BLOCKSIZE, BLOCKSIZE, 1);
    NMS_GPU<<<gridSize, blockSize>>>(d_b,d_res);
    
    hipDeviceSynchronize();

    gpuErrchk(hipMemcpy(h_res, d_res, sizeof(bool)*count, hipMemcpyDeviceToHost));

    //printf("Rest box:\n");
    for(int i =0; i<count ; i++)
    {

        if(*(h_res+i) != true)
        {
            //printf("%d ",i);
        } else{
            printf("%f %f %f %f\n", b[i].x, b[i].y, b[i].w, b[i].h);
            //printf("%f", b[i].x);
            //printf("%f", *(d_b+i)->x);
        }
    }
    //printf("\n");

    return 0;
}
