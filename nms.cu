#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include <stdbool.h>
using namespace cv;
using namespace std;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


typedef struct
{
	float x,y,w,h,s;

}box;

__device__
float IOUcalc(box b1, box b2)
{
	float ai = (float)(b1.w + 1)*(b1.h + 1);
	float aj = (float)(b2.w + 1)*(b2.h + 1);
	float x_inter, x2_inter, y_inter, y2_inter;

	x_inter = max(b1.x,b1.x);
	y_inter = max(b1.y,b2.y);
	
	x2_inter = min((b1.x + b1.w),(b2.x + b2.w));
	y2_inter = min((b1.y + b1.h),(b2.y + b2.h));
	
	float w = (float)max((float)0, x2_inter - x_inter);  
	float h = (float)max((float)0, y2_inter - y_inter);  
	
	float inter = ((w*h)/(ai + aj - w*h));
	return inter;
}



__global__
void NMS_GPU(box *d_b, int count, bool *d_res)
{
	//int abs_y = (blockIdx.y * blockDim.y) + threadIdx.y;
	//int abs_x = (blockIdx.x * blockDim.x) +threadId.x;
	int Id = (blockIdx.x * blockDim.x) + threadIdx.x;

    
    float theta = 0.6;
   	printf("\n%d--%f\n",d_res[Id]?1:0,d_b[Id].x);
    
    for(int i = 0;i<count; i++)
    {
    	
    	if(d_b[i].s < d_b[Id].s)
    	{
    		if(IOUcalc(d_b[i],d_b[Id])>theta)
	 		{
	 			d_res[i] = false; 
	 		}
    	}
    }
}





int main()
{
	int count =3;
	Mat temp = imread("/home/jeetkanjani7/crop001025b.png",1);
	
	bool *h_res =(bool *)malloc(sizeof(bool)*count);
	
	for(int i=0; i<count; i++)
	{
		h_res[i] = true;
	}
	
	
	box b[3];
	b[1].x = 16; b[1].y = 12; b[1].w = 64; b[1].h = 128; b[1].s = 0.79062;
	b[2].x = 12; b[2].y = 14; b[2].w = 70; b[2].h = 141; b[2].s = 0.60434 ;
	b[0].x = 11; b[0].y = 6; b[0].w = 74; b[0].h = 148; b[0].s = 0.11855;

	
	box *d_b;
	bool *d_res;
	
	gpuErrchk(hipMalloc((void**)&d_res, count*sizeof(bool)));
	gpuErrchk(hipMemcpy(d_res, h_res,sizeof(bool)*count, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc((void**)&d_b,sizeof(box)*count));
	gpuErrchk(hipMemcpy(d_b, b,sizeof(box)*count, hipMemcpyHostToDevice));
		
	NMS_GPU<<<1,count>>>(d_b,count,d_res);
	hipDeviceSynchronize();
	
	gpuErrchk(hipMemcpy(h_res, d_res, sizeof(bool)*count, hipMemcpyDeviceToHost));
	
	for(int i =0; i<3 ; i++)
	{
		printf("\nResults= %d--%d ",i,*(h_res+i));
		if(*(h_res+i) == true)
		{
			printf("Results= %d--%d ",i,*(h_res+i));
			rectangle(temp,Point(b[i].x,b[i].y),Point(b[i].x + b[i].w,b[i].y + b[i].h),Scalar(100,100,100),2,8,0);
		}
	}

	imshow("hello",temp);
	waitKey(0);
	return 0;
}
