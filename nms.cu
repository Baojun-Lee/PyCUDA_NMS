#include "hip/hip_runtime.h"
#include <stdio.h>
#include <hip/hip_runtime.h>
#include <iostream>
#include <opencv2/core/core.hpp>
#include <opencv2/highgui/highgui.hpp>
#include "opencv2/imgproc/imgproc.hpp"
#include <stdbool.h>
using namespace cv;
using namespace std;


#define gpuErrchk(ans) { gpuAssert((ans), __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char *file, int line, bool abort=true)
{
   if (code != hipSuccess) 
   {
      fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
      if (abort) exit(code);
   }
}


typedef struct
{
	float x,y,w,h,s;

}box;


__global__
void NMS_GPU(box *d_b, int count, bool *d_res)
{
	//int abs_y = (blockIdx.y * blockDim.y) + threadIdx.y;
	//int abs_x = (blockIdx.x * blockDim.x) +threadId.x;
	int Id = (blockIdx.x * blockDim.x) + threadIdx.x;

    float w,h,a;
    float theta = 0.6;
   	printf("\n%d--%f\n",Id,d_b[Id].x);
    
    for(int i = 0;i<count; i++)
    {
    	
    	if(d_b[i].s<d_b[Id].s)
    	{
    		a = (float)(d_b[Id].x + 1)*(d_b[Id].y + 1);
	 		w = (float)max((float)0,(min((d_b[i].x + d_b[i].w),(d_b[Id].x + d_b[Id].w)) - max(d_b[i].x,d_b[Id].x)));  
	 		h = (float)max((float)0,min((d_b[i].y + d_b[i].h),(d_b[Id].y + d_b[Id].h)) - max(d_b[i].y,d_b[Id].y));  
	 		if((w*h)/a > theta && d_b[Id].w > 0)
	 		{
	 			d_res[i] = false; 
	 		}
    	}
    }
}



int main()
{
	Mat temp = imread("/home/jeetkanjani7/crop001025b.png",1);
	
	int count =3;
	bool *h_res =(bool *)malloc(sizeof(bool)*count);
	for(int i=0; i<count; i++)
	{
		h_res[i] = true;
	}
	
	
	box b[3];
	b[1].x = 16; b[1].y = 12; b[1].w = 64; b[1].h = 128; b[1].s = 0.79062;
	b[2].x = 12; b[2].y = 14; b[2].w = 70; b[2].h = 141; b[2].s = 0.60434 ;
	b[0].x = 11; b[0].y = 6; b[0].w = 74; b[0].h = 148; b[0].s = 0.11855;

	printf("%d",(int)b[0].x);
	box *d_b;
	bool *d_res;
	
	gpuErrchk(hipMalloc((void**)&d_res, count*sizeof(bool)));
	gpuErrchk(hipMemcpy(d_res, h_res,sizeof(bool)*count, hipMemcpyHostToDevice));

	gpuErrchk(hipMalloc((void**)&d_b,sizeof(box)*count));
	gpuErrchk(hipMemcpy(d_b, b,sizeof(box)*count, hipMemcpyHostToDevice));
	
	NMS_GPU<<<1,count>>>(b,count,d_res);
	hipDeviceSynchronize();
	
	gpuErrchk(hipMemcpy(h_res, d_res, sizeof(bool)*count, hipMemcpyDeviceToHost));
	
	for(int i =0; i<3 ; i++)
	{
		printf("\nResults= %d--%d ",i,*(h_res+i));
		if(*(h_res+i) == true)
		{
			printf("Results= %d--%d ",i,*(h_res+i));
			rectangle(temp,Point(b[i].x,b[i].y),Point(b[i].x + b[i].w,b[i].y + b[i].h),Scalar(100,100,100),2,8,0);
		}
	}

	imshow("hello",temp);
	waitKey(0);
	return 0;
}
